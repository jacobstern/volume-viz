#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <hip/hip_vector_types.h>

#include <assert.h>
#include <stdio.h>

#include "kernel.cuh"
#include "assert.h"
#include "params.h"

#include <iostream>

#include "implicit.cu"

using std::cout;
using std::endl;

// TODO: Don't hardcode this
#define STEP_SIZE 0.00390625f // 1/256

#define CACHE_DEPTH             64
#define CACHE_DEPTH_MINUS_TWOF  62.f

#define DIRECT_FACTOR           0.3f
#define ONE_MINUS_DIRECT_FACTOR 0.7f

static struct hipGraphicsResource *pixelBuffer, *texture0, *texture1;

typedef texture<uchar4, hipTextureType2D, hipReadModeElementType> inTexture2D;
inTexture2D inTexture0, inTexture1;

typedef unsigned char uchar;

// volumetric texture
texture<unsigned char, hipTextureType3D, hipReadModeNormalizedFloat> texVolume;

hipArray *devVolume = 0;

__device__
float vectorLength(float3 vec)
{
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__
float ucharToFloat(unsigned char c)
{
    return c / 255.f;
}

__device__
bool boundsCheck(float3 pos)
{
    return pos.x < 1.0f && pos.x >= 0.0f
            && pos.y < 1.0f && pos.y >= 0.0f
            && pos.z < 1.0f && pos.z >= 0.0f;
}

__device__ unsigned char getVoxel(unsigned char sharedMemory[], dim3 cacheIdx, dim3 cacheDim, int offset) {
    if (offset < 0 || offset + 1 > CACHE_DEPTH)
        return 0x00;

    return sharedMemory[ offset * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ];
}

__device__
float blockMin(float shared[], int idx, int upper, float poll)
{
    shared[idx] = poll;

    __syncthreads();

    float min = shared[idx];

    for (int i = 0; i < upper; i++) {
        float compare = shared[i];
        if (compare < min) {
            min = compare;
        }
    }

    return min;
}

__device__
unsigned char sample(float3 pos) {
    return 0xff * tex3D(texVolume, pos.x, pos.y, pos.z);
}

__device__
float4 blend(float4 src, float4 dst) {
    float4 ret;
    float blendFactor = src.w * (1.f - dst.w);

    ret.x = dst.x + src.x * blendFactor;
    ret.y = dst.y + src.y * blendFactor;
    ret.z = dst.z + src.z * blendFactor;
    ret.w = dst.w +         blendFactor;

    return ret;
}

__device__
float4 transferFunction(uchar sampled) {
    float asFloat = ucharToFloat( sampled );

    return make_float4( asFloat, asFloat, asFloat, clamp(asFloat * asFloat * 2.f, 0.f, 1.f) );
}

__device__
void rayMarch(unsigned char cache[],
              dim3   cacheIdx,
              dim3   cacheDim,
              float3 origin,
              float3 direction) {
    float3 pos  = origin,
           step = direction * STEP_SIZE;

    for (int i = 0; i < CACHE_DEPTH; ++i) {
        cache[ i * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ]
                = sample( pos );

        pos += step;
    }

    __syncthreads();
}

#define DEBUG_PHONG

__device__
float4 shadeVoxel(unsigned char sharedMemory[],
                  dim3 cacheIdx,
                  dim3 cacheDim,
                  int offset,
                  float3 voxelDim) {
    uchar sampled
             = getVoxel( sharedMemory, cacheIdx, cacheDim, offset );

    float4 value = transferFunction( sampled );

#ifdef DEBUG_PHONG
    if ( value.w > 1e-6 ) {
        float l, r, t, b, f, a;

        f = ucharToFloat( getVoxel(sharedMemory, cacheIdx, cacheDim, offset - 1) );
        a = ucharToFloat( getVoxel(sharedMemory, cacheIdx, cacheDim, offset + 1) );

        l = ucharToFloat( getVoxel(sharedMemory, dim3(cacheIdx.x - 1, cacheIdx.y), cacheDim, offset) );
        r = ucharToFloat( getVoxel(sharedMemory, dim3(cacheIdx.x + 1, cacheIdx.y), cacheDim, offset) );
        t = ucharToFloat( getVoxel(sharedMemory, dim3(cacheIdx.x, cacheIdx.y + 1), cacheDim, offset) );
        b = ucharToFloat( getVoxel(sharedMemory, dim3(cacheIdx.x, cacheIdx.y - 1), cacheDim, offset) );

        float3 gradient = make_float3(
                    (r - l) / voxelDim.x,
                    (t - b) / voxelDim.y,
                    (a - f) / voxelDim.z );

        if (gradient.x != 0.f && gradient.y != 0.f&& gradient.z != 0.f)
            gradient = normalize(gradient);

        float  direct = dot( gradient, make_float3( -1.f, -1.f, 1.f ) ) * DIRECT_FACTOR;
        direct        = clamp(direct, 0.f, DIRECT_FACTOR);

        value =  make_float4( value.x * ONE_MINUS_DIRECT_FACTOR,
                              value.y * ONE_MINUS_DIRECT_FACTOR,
                              value.z * ONE_MINUS_DIRECT_FACTOR,
                              value.w );
        value += make_float4( direct, direct, direct, 0.f );
    }

#endif

    return value;
}

__device__
void mainLoop(uchar cache[],
              dim3 cacheIdx,
              dim3 cacheDim,
              dim3 imageDim,
              camera_params camera,
              float3 origin,
              float3 direction,
              float upper,
              float4 & result)
{
    float  dist = 0.f;
    result = make_float4( 0.f, 0.f, 0.f, 0.f );

    float  tanFovX = tan( camera.fovX * M_PI / (180.f * imageDim.x ) ),
           tanFovY = tan( camera.fovY * M_PI / (180.f * imageDim.y ) );

    while ( dist < upper ) { // No infinite loop plz
        float3 pos = origin + direction * dist;

        rayMarch( cache, cacheIdx, cacheDim, pos, direction );

        for (int i = 1; i < CACHE_DEPTH - 1; ++i) {
            float3 voxelDim = make_float3(
                        tanFovX * ( i * STEP_SIZE + dist ),
                        tanFovY * ( i * STEP_SIZE + dist ),
                        STEP_SIZE * 2.f
                        );
            float4 shaded = shadeVoxel( cache, cacheIdx, cacheDim, i, voxelDim );

            if (shaded.w > 1e-6) {
                result = blend( shaded, result );
            }

            if (result.w > .95f) {
                return;
            }
        }

        dist += STEP_SIZE * CACHE_DEPTH_MINUS_TWOF;
    }
}

#define SQRT_3 1.73205081f

__global__
void kernel(void *buffer,
            int width,
            int height,
            struct camera_params camera )
{
    extern __shared__ unsigned char sharedMemory[];
    uchar4 *pixels = (uchar4*) buffer;

    int x = blockIdx.x * (blockDim.x - 2) + ( ( (int) threadIdx.x ) - 1),
        y = blockIdx.y * (blockDim.y - 2) + ( ( (int) threadIdx.y ) - 1);

    int slabUpperX = min( (blockIdx.x + 1) * (blockDim.x - 2) + 1, width - 1  ),
        slabUpperY = min( (blockIdx.y + 1) * (blockDim.y - 2) + 1, height - 1 ),
        slabLowerX = max( (int) (blockIdx.x  * (blockDim.x - 2)) - 1, 0),
        slabLowerY = max( (int) (blockIdx.y  * (blockDim.y - 2)) - 1, 0),
        slabWidth  = slabUpperX - slabLowerX,
        slabHeight = slabUpperY - slabLowerY;

    bool isBorder = threadIdx.x == 0 || threadIdx.y == 0
            || threadIdx.x + 1 == blockDim.x || threadIdx.y + 1 == blockDim.y;

    x = clamp(x, slabLowerX, slabUpperX - 1);
    y = clamp(y, slabLowerY, slabUpperY - 1);

    int index = y * width + x;

    int slabY = y - slabLowerY,
        slabX = x - slabLowerX,
        slabIndex = slabY * slabWidth + slabX,
        slabUpper = slabHeight * slabWidth;

    uchar4 sample0 = tex2D( inTexture0, (float) x / width, (float) y / height ),
           sample1 = tex2D( inTexture1, (float) x / width, (float) y / height );

    float3 front = make_float3(sample0.x / 255.f, sample0.y / 255.f, sample0.z / 255.f),
           back  = make_float3(sample1.x / 255.f, sample1.y / 255.f, sample1.z / 255.f);

    float3  camPos  = make_float3( camera.origin[0], camera.origin[1], camera.origin[2] ),
            camDist = front - camPos;
    float camLength = vectorLength(camDist);

    // Note: all threads in block where index < width * height
    // MUST execute this function. Also NB this includes a thread barrier.
    float rad = blockMin((float*)sharedMemory, slabIndex, slabUpper, camLength);

    float3 dist = back - front;
    float length = vectorLength(dist);

    if (length < 0.001f && !isBorder) {
        // TODO: set a better min value if this happens
        pixels[index] = make_uchar4(0, 0, 0, 0);
        return;
    }

    float3 ray   = dist / length,
           pos   = front;

    float t;
    bool success = intersectSphereAndRay(camPos, rad, front, -ray, t);
    if (success) {
        // Update front based on desired distance from camera
        pos = pos - ray * t;
    }

    float upper = fminf( SQRT_3, vectorLength( back - pos ) );

    dim3 cacheIdx(slabX, slabY),
         cacheDim(slabWidth, slabHeight),
         imageDim(width, height);

    float4 result;
    mainLoop(sharedMemory, cacheIdx, cacheDim, imageDim, camera, pos, ray, upper, result);

    if (!isBorder) {
        result.x = clamp(result.x, 0.f, 1.f);
        result.y = clamp(result.y, 0.f, 1.f);
        result.z = clamp(result.z, 0.f, 1.f);
        result.w = clamp(result.w, 0.f, 1.f);

        pixels[index] = make_uchar4(result.x * 0xff, result.y * 0xff, result.z * 0xff, result.w * 0xff);
    }
}

void initCuda() {
    cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );

    SDK_CHECK_ERROR_GL();
}

void registerCudaResources(GLuint input0, GLuint input1, GLuint output) {
    assert(input0);
    assert(input1);
    assert(output);

    checkCudaErrors( hipGraphicsGLRegisterImage(&texture0, input0, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly) );
    checkCudaErrors( hipGraphicsGLRegisterImage(&texture1, input1, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly) );
    checkCudaErrors( hipGraphicsGLRegisterBuffer(&pixelBuffer, output, hipGraphicsRegisterFlagsWriteDiscard) );

    inTexture0.normalized = true;
    inTexture1.normalized = true;
}

void runCuda(int width,
             int height,
             struct slice_params slice,
             struct camera_params camera,
             struct shading_params shading,
             hipArray* volumeArray) {
    hipGraphicsResource_t resources[3] = { texture0, texture1, pixelBuffer };

    checkCudaErrors( hipGraphicsMapResources(3, resources) );

    struct hipArray *array0;
    checkCudaErrors( hipGraphicsSubResourceGetMappedArray(&array0, texture0, 0, 0) );
    checkCudaErrors( hipBindTextureToArray(inTexture0, array0) );

    struct hipArray *array1;
    checkCudaErrors( hipGraphicsSubResourceGetMappedArray(&array1, texture1, 0, 0) );
    checkCudaErrors( hipBindTextureToArray(inTexture1, array1) );

    void *devBuffer;
    size_t bufferSize;
    checkCudaErrors( hipGraphicsResourceGetMappedPointer(&devBuffer, &bufferSize, pixelBuffer) );

    // For convenience, greedily chunk the screen into 14x14 squares
    dim3 blockSize(14, 14),
         blockDims(width / blockSize.x, height / blockSize.y);
    if (width % blockSize.x)
        ++blockDims.x;
    if (height % blockSize.y)
        ++blockDims.y;

    blockSize.x += 2;
    blockSize.y += 2;

    size_t sharedMemSize = ( CACHE_DEPTH ) * ( blockSize.x ) * ( blockSize.y ) * sizeof( uchar );
    kernel<<< blockDims, blockSize, sharedMemSize >>>(devBuffer, width, height, camera);

    checkCudaErrors( hipUnbindTexture(inTexture0) );

    checkCudaErrors( hipGraphicsUnmapResources(3, resources) );
}

// load volumetric texture into the GPU
void cudaLoadVolume(byte* texels, size_t size, Vector3 dims,
                    hipArray** volumeArray) {

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();

    cout << "mallocing texture array" << endl;

    hipExtent extent = make_hipExtent( dims.x, dims.y, dims.z );
    checkCudaErrors( hipMalloc3DArray(&devVolume, &channelDesc, extent) );

    assert(texels);
    assert(devVolume);
    assert(size);

    int width = dims.x;
    int height = dims.y;
    int depth = dims.z;

    hipMemcpy3DParms params = {0};
    params.srcPtr = make_hipPitchedPtr(texels, width * sizeof(unsigned char), width, height);
    params.dstArray = devVolume;
    params.extent = make_hipExtent(width, height, depth);
    params.kind = hipMemcpyHostToDevice;

    checkCudaErrors( hipMemcpy3D(&params) );

    // set addressmode
    texVolume.normalized = true;
    texVolume.filterMode = hipFilterModeLinear;
    texVolume.addressMode[0] = hipAddressModeClamp;
    texVolume.addressMode[1] = hipAddressModeClamp;
    texVolume.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray(texVolume, devVolume, channelDesc));


}






