#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <hip/hip_vector_types.h>

#include <assert.h>
#include <stdio.h>

#include "kernel.cuh"
#include "implicit.cu"

static struct hipGraphicsResource *pixelBuffer, *texture0, *texture1;

typedef texture<uchar4, hipTextureType2D, hipReadModeElementType> inTexture2D;
inTexture2D inTexture0, inTexture1;

__device__
float vectorLength(float3 vec)
{
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__
bool boundsCheck(float3 pos)
{
    return pos.x < 1.0f && pos.x >= 0.0f
            && pos.y < 1.0f && pos.y >= 0.0f
            && pos.z < 1.0f && pos.z >= 0.0f;
}

__device__
float blockMin(float shared[], int idx, int upper, float poll)
{
    shared[idx] = poll;

    __syncthreads();

    float min = shared[idx];

    for (int i = 0; i < upper; i++) {
        float compare = shared[i];
        if (compare < min) {
            min = compare;
        }
    }

    return min;
}

__device__
float4 sampleVolume(float3 pos)
{
    // TODO: Sample from volume texture
    if (pos.x > 1.f || pos.y > 1.f || pos.z > 1.f
            || pos.x < 0.f || pos.y < 0.f || pos.z < 0.f) {
        return make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    if ((pos.x - .5f) * (pos.x - .5f) + (pos.z - .5f) * (pos.z - .5f) < .25) {
       return make_float4(1.f, 1.f, 1.f, 0.01f);
    }
    else {
       return make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }
}

#define MAX_STEPS 63

__device__
unsigned char sample(float3 pos) {
    if ( boundsCheck(pos) && (pos.x - .5f) * (pos.x - .5f) + (pos.z - .5f) * (pos.z - .5f) < .25 ) {
        return 0xff;
    }

    return 0x00;
}

__device__
void rayMarch(unsigned char sharedMemory[], float3 origin, float3 step, dim3 cacheIdx, dim3 cacheDim, int lower=0, int upper=MAX_STEPS) {
    float3 pos = origin + lower * step;
    unsigned char i = 0x00;

    for (; i < lower; ++i)
        sharedMemory[ (i + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ]
                = 0x00;

    for (; i < upper; ++i) {
        if (pos.x < 1.0f && pos.x >= 0.0f
                && pos.y < 1.0f && pos.y >= 0.0f
                && pos.z < 1.0f && pos.z >= 0.0f) {
            break;
        }

        sharedMemory[ (i + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ]
                = sample(pos);

        pos += step;
    }


    for (; i < upper; ++i) {
        if (pos.x >= 1.0f || pos.x < 0.0f
                || pos.y >= 1.0f || pos.y < 0.0f
                || pos.z >= 1.0f || pos.z < 0.0f) {
            break;
        }

        sharedMemory[ (i + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ]
                = sample(pos);

        pos += step;
    }

    sharedMemory[ cacheIdx.y * cacheDim.x + cacheIdx.x ] = i;

    // Fill up the rest of the cache with zeros

    for (; i < MAX_STEPS; ++i)
        sharedMemory[ (i + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ]
                = 0x00;

    __syncthreads();
}

__device__
float4 shadePhong(unsigned char sharedMemory[], dim3 cacheIdx, dim3 cacheDim) {
    unsigned char upper = sharedMemory[ cacheIdx.y * cacheDim.x + cacheIdx.x ];

    for (unsigned char i = 0; i < upper; ++i) {
        unsigned char sampled =  sharedMemory[ (i + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ];

        if (sampled) {
            return make_float4(1.f, 1.f, 1.f, 1.f);
        }
    }

    return make_float4(0.f, 0.f, 0.f, 0.f);
}

__device__ unsigned char getVoxel(unsigned char sharedMemory[], dim3 cacheIdx, dim3 cacheDim, int offset) {
    if (offset < 0 || offset + 1 > MAX_STEPS)
        return 0x00;

    return sharedMemory[ (offset + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ];
}

#define DEBUG_TRANSPARENT

__device__
float4 shadeVoxel(unsigned char sharedMemory[], dim3 cacheIdx, dim3 cacheDim, int offset, float stepSize) {
#ifdef DEBUG_TRANSPARENT
    unsigned char sampled =  sharedMemory[ (offset + 1) * cacheDim.x * cacheDim.y + cacheIdx.y * cacheDim.x + cacheIdx.x ];

    return make_float4(sampled / 255.f, sampled / 255.f, sampled / 255.f, sampled * 0.01f * stepSize / 255.f);
#endif

#ifdef DEBUG_PHONG
    unsigned char c, l, r, t, b, f, a;

    c = getVoxel(sharedMemory, cacheIdx, cacheDim, offset);
    f = getVoxel(sharedMemory, cacheIdx, cacheDim, offset - 1);
    a = getVoxel(sharedMemory, cacheIdx, cacheDim, offset + 1);

    l = getVoxel(sharedMemory, dim3(cacheIdx.x - 1, cacheIdx.y), cacheDim, offset);
    r = getVoxel(sharedMemory, dim3(cacheIdx.x + 1, cacheIdx.y), cacheDim, offset);
    t = getVoxel(sharedMemory, dim3(cacheIdx.x, cacheIdx.y + 1), cacheDim, offset);
    b = getVoxel(sharedMemory, dim3(cacheIdx.x, cacheIdx.y - 1), cacheDim, offset);


#endif

    return make_float4(0.f, 0.f, 0.f, 0.f);
}

__device__
float4 shade(unsigned char sharedMemory[], dim3 cacheIdx, dim3 cacheDim, float normalize) {
    unsigned char upper = sharedMemory[ cacheIdx.y * cacheDim.x + cacheIdx.x ];
    float4 accum = make_float4(0.f, 0.f, 0.f, 0.f);

    for (unsigned char i = 0; i < upper; ++i) {
        float4 vox = shadeVoxel(sharedMemory, cacheIdx, cacheDim, i, normalize);

        if (vox.w > 1e-6) {
            accum.x += vox.x * vox.w * (1.f - accum.w);
            accum.y += vox.y * vox.w * (1.f - accum.w);
            accum.z += vox.z * vox.w * (1.f - accum.w);
            accum.w += vox.w * (1.f - accum.w);

            if (accum.w > .95f) {
                break;
            }
        }
    }

    accum.x = fminf(accum.x, 1.f);
    accum.y = fminf(accum.y, 1.f);
    accum.z = fminf(accum.z, 1.f);
    accum.w = fminf(accum.w, 1.f);

    return accum;
}

#define SQRT_3 1.73205081f

__global__
void kernel(void *buffer,
            int width,
            int height,
            struct slice_params slice,
            struct camera_params camera )
{
    extern __shared__ unsigned char sharedMemory[];
    uchar4 *pixels = (uchar4*) buffer;

    int x = blockIdx.x * (blockDim.x - 2) + ( ( (int) threadIdx.x ) - 1),
        y = blockIdx.y * (blockDim.y - 2) + ( ( (int) threadIdx.y ) - 1);

    int slabUpperX = min( (blockIdx.x + 1) * (blockDim.x - 2) + 1, width - 1  ),
        slabUpperY = min( (blockIdx.y + 1) * (blockDim.y - 2) + 1, height - 1 ),
        slabLowerX = max( (int) (blockIdx.x  * (blockDim.x - 2)) - 1, 0),
        slabLowerY = max( (int) (blockIdx.y  * (blockDim.y - 2)) - 1, 0),
        slabWidth  = slabUpperX - slabLowerX,
        slabHeight = slabUpperY - slabLowerY;

    bool isBorder = threadIdx.x == 0 || threadIdx.y == 0
            || threadIdx.x + 1 == blockDim.x || threadIdx.y + 1 == blockDim.y;

    x = clamp(x, slabLowerX, slabUpperX - 1);
    y = clamp(y, slabLowerY, slabUpperY - 1);

    int index = y * width + x;

    int slabY = y - slabLowerY,
        slabX = x - slabLowerX,
        slabIndex = slabY * slabWidth + slabX,
        slabUpper = slabHeight * slabWidth;

    uchar4 sample0 = tex2D( inTexture0, x, y ),
           sample1 = tex2D( inTexture1, x, y );

    float3 front = make_float3(sample0.x / 255.f, sample0.y / 255.f, sample0.z / 255.f),
           back  = make_float3(sample1.x / 255.f, sample1.y / 255.f, sample1.z / 255.f);

    float3  camPos  = make_float3( camera.origin[0], camera.origin[1], camera.origin[2] ),
            camDist = front - camPos;
    float camLength = vectorLength(camDist);

    // Note: all threads in block where index < width * height
    // MUST execute this function. Also NB this includes a thread barrier.
    float rad = blockMin((float*)sharedMemory, slabIndex, slabUpper, camLength);

    float3 dist = back - front;
    float length = vectorLength(dist);

    if (length < 0.001f && !isBorder) {
        // TODO: set a better min value if this happens
        pixels[index] = make_uchar4(0, 0, 0, 0);
        return;
    }

    float3 ray   = dist / length,
           pos   = front;

    if (slice.type == SLICE_PLANE) {
        float3 point  = make_float3( slice.params[0], slice.params[1], slice.params[2] ),
               normal = make_float3( slice.params[3], slice.params[4], slice.params[5] );

        // TODO: Slicing
    }

    float t;
    bool success = intersectSphereAndRay(camPos, rad, front, -ray, t);
    if (success) {
        // Update front based on desired distance from camera
        pos = pos - ray * t;
    }

    float  distActual = vectorLength(back - pos),
           stepSize = fminf( SQRT_3, distActual ) / MAX_STEPS;

    float3 step = ray * stepSize;

    dim3 cacheIdx(slabX, slabY),
         cacheDim(slabWidth, slabHeight);

    rayMarch(sharedMemory, pos, step, cacheIdx, cacheDim);

    if (!isBorder) {
        float4 shaded = shade(sharedMemory, cacheIdx, cacheDim, stepSize);

        pixels[index] = make_uchar4(shaded.x * 0xff, shaded.y * 0xff, shaded.z * 0xff, shaded.w * 0xff);
    }
}

void initCuda() {
    cudaGLSetGLDevice( gpuGetMaxGflopsDeviceId() );

    SDK_CHECK_ERROR_GL();
}

void registerCudaResources(GLuint input0, GLuint input1, GLuint output) {
    assert(input0);
    assert(input1);
    assert(output);

    checkCudaErrors( hipGraphicsGLRegisterImage(&texture0, input0, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly) );
    checkCudaErrors( hipGraphicsGLRegisterImage(&texture1, input1, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly) );
    checkCudaErrors( hipGraphicsGLRegisterBuffer(&pixelBuffer, output, hipGraphicsRegisterFlagsWriteDiscard) );
}

void runCuda(int width, int height, struct slice_params slice, struct camera_params camera) {
    hipGraphicsResource_t resources[3] = { texture0, texture1, pixelBuffer };

    checkCudaErrors( hipGraphicsMapResources(3, resources) );

    struct hipArray *array0;
    checkCudaErrors( hipGraphicsSubResourceGetMappedArray(&array0, texture0, 0, 0) );
    checkCudaErrors( hipBindTextureToArray(inTexture0, array0) );

    struct hipArray *array1;
    checkCudaErrors( hipGraphicsSubResourceGetMappedArray(&array1, texture1, 0, 0) );
    checkCudaErrors( hipBindTextureToArray(inTexture1, array1) );

    void *devBuffer;
    size_t bufferSize;
    checkCudaErrors( hipGraphicsResourceGetMappedPointer(&devBuffer, &bufferSize, pixelBuffer) );

    // For convenience, greedily chunk the screen into 256-pixel squares
    dim3 blockSize(14, 14),
         blockDims(width / blockSize.x, height / blockSize.y);
    if (width % blockSize.x)
        ++blockDims.x;
    if (height % blockSize.y)
        ++blockDims.y;

    blockSize.x += 2;
    blockSize.y += 2;

    size_t sharedMemSize = ( MAX_STEPS + 1 ) * ( blockSize.x ) * ( blockSize.y ) * sizeof( unsigned char );
    kernel<<< blockDims, blockSize, sharedMemSize >>>(devBuffer, width, height, slice, camera);

    checkCudaErrors( hipUnbindTexture(inTexture0) );

    checkCudaErrors( hipGraphicsUnmapResources(3, resources) );
}
